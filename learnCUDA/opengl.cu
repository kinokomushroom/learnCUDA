#include "hip/hip_runtime.h"
#include <glad/glad.h>
#include <GLFW/glfw3.h>
#include <glm/glm.hpp>
#include <glm/gtc/matrix_transform.hpp>
#include <glm/gtc/type_ptr.hpp>

#include <iostream>
#include <cmath>

#include <hip/hip_runtime.h>
#include <>
#include <cuda_gl_interop.h>

#include "stb_image.h"

#include "shader.h"
#include "curved_geometry.cuh"

GLFWwindow* initOpenGL();
void framebufferSizeCallback(GLFWwindow* window, int width, int height);
void processInput(GLFWwindow* window);
unsigned int createTexture(int textureSize_x, int textureSize_y);
unsigned int createQuadVAO();
void renderTextureCUDA(hipGraphicsResource_t textureResource, double* coords, int textureSize_x, int textureSize_y);


const unsigned int SCREEN_WIDTH = 256;
const unsigned int SCREEN_HEIGHT = 256;
const unsigned int TEXTURE_SIZE = 256;

double deltaTime = 0.0;
double lastTime = 0.0;

bool updateFrame = true;

const double MOVE_DISTANCE = 0.1;
int input[2] = { 0, 0 };
double initialPosition[2] = { 1.0, 1.0 };
double position[2];
double basis[4]; // basis[a, b] = basis[2 * a + b];

int displayMode = LINES;
bool displayKeyPressed = false;


int main()
{
	// initialize GLFW and GLAD, and create a window
	GLFWwindow* window;
	window = initOpenGL();
	if (window == NULL)
	{
		return -1;
	}

	// set callbacks
	glfwSetFramebufferSizeCallback(window, framebufferSizeCallback);

	// create texture to display
	unsigned int texture = createTexture(TEXTURE_SIZE, TEXTURE_SIZE);

	// register texture with CUDA
	hipGraphicsResource_t textureResource;
	hipGraphicsGLRegisterImage(&textureResource, texture, GL_TEXTURE_2D, hipGraphicsRegisterFlagsSurfaceLoadStore);
	
	// register texture to shader
	glActiveTexture(GL_TEXTURE0);
	glBindTexture(GL_TEXTURE_2D, texture);

	// set up VAO and VBO
	unsigned int VAO = createQuadVAO();

	// set up shader
	Shader shader = Shader("vertex.glsl", "fragment.glsl");


	double* coords;
	size_t bytes = 2 * TEXTURE_SIZE * TEXTURE_SIZE * sizeof(double);
	hipMalloc(&coords, bytes);

	// initialize position and basis
	position[0] = initialPosition[0];
	position[1] = initialPosition[1];
	initializeBasis(position, basis);
	//printArray(basis, 4, "basis");


	// render loop
	while (!glfwWindowShouldClose(window))
	{
		// calculate delta time
		double currentTime = glfwGetTime();
		deltaTime = currentTime - lastTime;
		lastTime = currentTime;
		//if (floor(currentTime - deltaTime) < floor(currentTime))
		//{
		//	std::cout << (int)floor(currentTime) << ": " << deltaTime << std::endl;
		//}

		processInput(window);

		glClearColor(0.0f, 0.0f, 0.0f, 1.0f);
		glClear(GL_COLOR_BUFFER_BIT);

		if (updateFrame)
		{
			updateFrame = false;

			// measure time
			double startTime = glfwGetTime();

			// update coords
			unsigned int blockSize = 16;
			dim3 blockDimension(blockSize, blockSize);
			dim3 gridDimension((int)ceil((float)TEXTURE_SIZE / blockSize), (int)ceil((float)TEXTURE_SIZE / blockSize));
			calculateCoords<<<gridDimension, blockDimension>>>(coords, TEXTURE_SIZE, TEXTURE_SIZE, 4.0, 4.0, basis[0], basis[1], basis[2], basis[3], position[0], position[1]);
			hipDeviceSynchronize();

			// render texture with CUDA
			renderTextureCUDA(textureResource, coords, TEXTURE_SIZE, TEXTURE_SIZE);

			// output position
			//std::cout << "x: " << position[0] << ", y: " << position[1] << std::endl;
			//printArray(basis, 4, "basis");

			// output elapsed time
			double elapsedTime = glfwGetTime() - startTime;
			//std::cout << elapsedTime << " sec" << std::endl;
		}

		// draw texture
		shader.use();
		glBindVertexArray(VAO);
		glDrawArrays(GL_TRIANGLE_STRIP, 0, 4);

		glfwSwapBuffers(window);
		glfwPollEvents();
	}

	hipFree(coords);

	glfwTerminate();
	return 0;
}


GLFWwindow* initOpenGL()
{
	// initialize GLFW
	glfwInit();
	glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 4); // use OpenGL version 4.6
	glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 6);
	glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);

	// create a window
	GLFWwindow* window = glfwCreateWindow(SCREEN_WIDTH, SCREEN_HEIGHT, "LearnOpenGL", NULL, NULL);
	if (window == NULL)
	{
		std::cout << "Failed to create GLFW window" << std::endl;
		glfwTerminate();
		return NULL;
	}
	glfwMakeContextCurrent(window);

	// initialize GLAD
	if (!gladLoadGLLoader((GLADloadproc)glfwGetProcAddress))
	{
		std::cout << "Failed to initialize GLAD" << std::endl;
		return NULL;
	}
	return window;
}


void framebufferSizeCallback(GLFWwindow* window, int width, int height)
{
	glViewport(0, 0, width, height);
}


void processInput(GLFWwindow* window)
{
	// handle window close
	if (glfwGetKey(window, GLFW_KEY_ESCAPE) == GLFW_PRESS)
	{
		glfwSetWindowShouldClose(window, true);
	}

	// change display mode
	if (glfwGetKey(window, GLFW_KEY_C) == GLFW_PRESS)
	{
		if (!displayKeyPressed)
		{
			displayKeyPressed = true;
			updateFrame = true;
			displayMode = (displayMode + 1) % 3;
			//std::cout << "display mode changed!" << std::endl;
		}
	}
	else
	{
		displayKeyPressed = false;
	}

	// handle movement input
	input[0] = 0;
	input[1] = 0;
	if (glfwGetKey(window, GLFW_KEY_D) == GLFW_PRESS)
	{
		input[0] += 1;
	}
	if (glfwGetKey(window, GLFW_KEY_A) == GLFW_PRESS)
	{
		input[0] -= 1;
	}
	if (glfwGetKey(window, GLFW_KEY_W) == GLFW_PRESS)
	{
		input[1] += 1;
	}
	if (glfwGetKey(window, GLFW_KEY_S) == GLFW_PRESS)
	{
		input[1] -= 1;
	}
	if (input[0] != 0 || input[1] != 0) // if not zero, update position and basis
	{
		updateFrame = true;
		updatePosition(position, basis, input, MOVE_DISTANCE);
		//printArray(basis, 4, "basis");
	}
}


unsigned int createTexture(int textureSize_x, int textureSize_y)
{
	unsigned int texture;
	glGenTextures(1, &texture);
	glBindTexture(GL_TEXTURE_2D, texture);
	{
		glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
		glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
		glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA32F, textureSize_x, textureSize_y, 0, GL_RGBA, GL_UNSIGNED_BYTE, NULL);
	}
	glBindTexture(GL_TEXTURE_2D, 0);
	return texture;
}


unsigned int createQuadVAO()
{
	// quad that fills screen
	float vertexData[] =
	{
		// position         uv
		-1.0, -1.0,  0.0,   0.0, 0.0,
		 1.0, -1.0,  0.0,   1.0, 0.0,
		-1.0,  1.0,  0.0,   0.0, 1.0,
		 1.0,  1.0,  0.0,   1.0, 1.0
	};

	unsigned int VAO;
	glGenVertexArrays(1, &VAO);
	unsigned int VBO;
	glGenBuffers(1, &VBO);
	glBindVertexArray(VAO);
	glBindBuffer(GL_ARRAY_BUFFER, VBO);
	glBufferData(GL_ARRAY_BUFFER, sizeof(vertexData), vertexData, GL_STATIC_DRAW);
	// set up vertex attribute pointers
	glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, 5 * sizeof(float), (void*)0);
	glVertexAttribPointer(1, 2, GL_FLOAT, GL_FALSE, 5 * sizeof(float), (void*)(3 * sizeof(float)));
	glEnableVertexAttribArray(0);
	glEnableVertexAttribArray(1);
	// unbind
	glBindVertexArray(0);
	glBindBuffer(GL_ARRAY_BUFFER, 0);

	return VAO;
}


void renderTextureCUDA(hipGraphicsResource_t textureResource, double* coords, int textureSize_x, int textureSize_y)
{
	hipGraphicsMapResources(1, &textureResource, 0);
	// get mapped array
	hipArray_t textureArray;
	hipGraphicsSubResourceGetMappedArray(&textureArray, textureResource, 0, 0);
	// specify surface
	struct hipResourceDesc resourceDescription;
	memset(&resourceDescription, 0, sizeof(resourceDescription));
	resourceDescription.resType = hipResourceTypeArray;
	// create surface object
	resourceDescription.res.array.array = textureArray;
	hipSurfaceObject_t surfaceObject;
	hipCreateSurfaceObject(&surfaceObject, &resourceDescription);
	// launch kernel
	unsigned int blockSize = 16;
	dim3 blockDimension(blockSize, blockSize);
	dim3 gridDimension((int)ceil((float)textureSize_x / blockSize), (int)ceil((float)textureSize_y / blockSize));
	renderTextureKernel<<<gridDimension, blockDimension>>>(surfaceObject, coords, TEXTURE_SIZE, TEXTURE_SIZE, displayMode);
	hipDeviceSynchronize();
	// free
	hipDestroySurfaceObject(surfaceObject);
	//hipFreeArray(textureArray); // DO NOT FREE ARRAY!!
	hipGraphicsUnmapResources(1, &textureResource, 0);
}